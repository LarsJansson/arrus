#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <array>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "hsdi.cuh"
#include "helper.h"

// SHAPE (NEVENTS, NCHANNELS_OX, NCHANNELS_OY, NSAMPLES)

const unsigned NEVENTS = 1;
const unsigned NCHANNELS_OX = 32;
const unsigned NCHANNELS_OY = 32;
const unsigned NSAMPLES = 2048;
const unsigned DATA_SIZE = NEVENTS*NCHANNELS_OX*NCHANNELS_OY*NSAMPLES;
std::array<dtype, DATA_SIZE> inputBuffer;

const unsigned PADDED_OX = NCHANNELS_OX*2;
const unsigned PADDED_OY = NCHANNELS_OY*2;
const unsigned PADDED_DATA_SIZE = NEVENTS*PADDED_OX*PADDED_OY*NSAMPLES;
std::array<dtype, PADDED_DATA_SIZE> outputBuffer;

int main(int argc, char* argv[])
{
    typedef double realType;
    typedef hipfftDoubleComplex complexType;
    // Read data
    realType *devInBuffer, *devProcBuffer, *devOutputBuffer;
    complexType *fftBuffer;
    hipfftHandle fftPlanFwd, fftPlanInv;

    std::ifstream input{"data.bin", std::ios::binary};
    input.read((char*)(inputBuffer.data()), inputBuffer.size()*sizeof(dtype));
    checkCudaErrors(hipMalloc(&devInBuffer, DATA_SIZE*sizeof(dtype)));
    checkCudaErrors(hipMemcpy(devInBuffer, inputBuffer.data(), DATA_SIZE*sizeof(dtype), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc(&devProcBuffer, PADDED_DATA_SIZE*sizeof(dtype)));
    checkCudaErrors(hipMalloc(&fftBuffer,
                               (PADDED_DATA_SIZE/2+1)*sizeof(complexType)));
    checkCudaErrors(hipMalloc(&devOutputBuffer,
                               (PADDED_DATA_SIZE)*sizeof(realType)));
    checkCudaErrors(hipfftPlan3d(&fftPlanFwd, PADDED_OX, PADDED_OY,
                                NSAMPLES, HIPFFT_D2Z));
    checkCudaErrors(hipfftPlan3d(&fftPlanInv, PADDED_OX, PADDED_OY,
                                NSAMPLES, HIPFFT_Z2D));


    // Pad with zeros.
    dim3 threads(32, 8, 1);
    dim3 grid(divup(PADDED_OX, threads.x),
              divup(PADDED_OY, threads.y),
              divup(NSAMPLES, threads.z));

    std::cout << "Padding with zeros" << std::endl;
    padHalfWithZeros<<<grid, threads>>>(devProcBuffer, devInBuffer,
                                        PADDED_OX, PADDED_OY,
                                        NCHANNELS_OX, NCHANNELS_OY, NSAMPLES);
    // FFT
    checkCudaErrors(hipfftExecD2Z(fftPlanFwd, devProcBuffer, fftBuffer));

    // Interpolation & weighting

    // IFFT
    checkCudaErrors(hipfftExecZ2D(fftPlanInv, fftBuffer, devOutputBuffer));

    // Abs, norm?

    // Write output to a file.
    std::cout << "Producing the output, size: "
              << PADDED_DATA_SIZE*sizeof(dtype)
              << std::endl;
    std::ofstream output{"pdata.bin", std::ios::binary};

    checkCudaErrors(hipMemcpy(outputBuffer.data(), devProcBuffer,
                               PADDED_DATA_SIZE*sizeof(dtype),
                               hipMemcpyDeviceToHost));

    output.write((char*)(outputBuffer.data()), outputBuffer.size()*sizeof(dtype));
    hipFree(devInBuffer);
    hipFree(devProcBuffer);
    hipfftDestroy(fftPlanFwd);
    hipfftDestroy(fftPlanInv);
    return 0;
}
